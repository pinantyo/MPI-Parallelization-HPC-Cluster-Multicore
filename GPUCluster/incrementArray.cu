#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

void incrementArrayOnHost(float* a, int N) {
	    for (int i = 0; i < N; i++) {
		            a[i] = a[i] + 1.f;
			        }
}

__global__ void incrementArrayOnDevice(float* a, int N) {
	    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	        if (idx < N) {
			        a[idx] = a[idx] + 1.f;
				    }
}

int main(void) {
	    int numTests = 13; // Number of tests with different N values
	        int NValues[] = {2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 2048, 4096, 4096}; // Array of N values to test
		    int blockSizes[] = {2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 1024, 1024, 2048};  //Corresponding blockSizes


		        for (int t = 0; t < numTests; t++) {
				        int N = NValues[t];
					        int blockSize = blockSizes[t]; // Use corresponding blockSize


						        float* a_h, * b_h;
							        float* a_d;
								        size_t size = N * sizeof(float);


									        printf("Running test with N = %d and blockSize = %d\n", N, blockSize);

										        // allocate arrays on host
										        a_h = (float*)malloc(size);
											        b_h = (float*)malloc(size);

												        // allocate array on device
												        hipMalloc((void**)&a_d, size);

													        // initialization of host data
													        for (int i = 0; i < N; i++) {
															            a_h[i] = (float)i;
																            }

														        // copy data from host to device
														        hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

															        // do calculation on host
															        incrementArrayOnHost(a_h, N);

																        // do calculation on device: compute execution configuration and launch
																        int nBlocks = (N + blockSize - 1) / blockSize;  // Ensure correct ceiling
																	        incrementArrayOnDevice<<<nBlocks, blockSize>>>(a_d, N);

																		        // Retrieve result from device and store in b_h
																		        hipMemcpy(b_h, a_d, size, hipMemcpyDeviceToHost);

																			        // check results
																			        for (int i = 0; i < N; i++) {
																					            assert(a_h[i] == b_h[i]);
																						            }

																				        printf("Test with N = %d and blockSize = %d passed.\n\n", N, blockSize);

																					        // cleanup
																					        free(a_h);
																						        free(b_h);
																							        hipFree(a_d);
																								    }  // End of test loop

			    return 0;
}
