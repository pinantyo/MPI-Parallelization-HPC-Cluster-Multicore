
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel ( int *a, int *b)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  a[idx] = threadIdx.x;
  b[idy] = threadIdx.y;
}


int main(void)
{
  int *a_h,*b_h;
  int *a_d,*b_d;
  int N=64;
  a_h = (int *)malloc(N*sizeof(int));
  b_h = (int *)malloc(N*sizeof(int));
  hipMalloc((void **) &a_d, N*sizeof(int));
  hipMalloc((void **) &b_d, N*sizeof(int));
  for(int i = 0; i<N; i++){
	  a_h[i] = 9;
	  b_h[i] = 8;
  }
  hipMemcpy(a_d, a_h, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, N*sizeof(int), hipMemcpyHostToDevice);
  //for (int i=0; i< 14; i++) assert( a_h[i] == b_h[i] );
  dim3 grid, block;
  grid.x = 3; grid.y = 3;
  block.x = 8; block.y = 16;
  kernel<<<grid,block>>>(a_d,b_d);
  hipMemcpy(a_h,a_d,N*sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(b_h,b_d,N*sizeof(int),hipMemcpyDeviceToHost);
  for(int i = 0; i < N; i++)
  {
	  printf("%d", a_h[i]);
  }
  printf("\n");
  for(int i = 0; i < N; i++){
	  printf("%d", b_h[i]);
  
  }
  hipDeviceSynchronize();
  free(a_h); free(b_h); hipFree(b_d); hipFree(a_d);
  return 0;
}

