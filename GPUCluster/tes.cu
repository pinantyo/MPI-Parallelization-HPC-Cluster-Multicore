
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* a) {
	    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	        a[idx] = 7;
}

int main() {
	    int N = 256;
	        int* a_h;
		    int* a_d;

		        a_h = (int*)malloc(N * sizeof(int));

			    hipMalloc((void**)&a_d, N * sizeof(int));

			        int blockSize = 256;
				    int gridSize = (N + blockSize - 1) / blockSize;

				        kernel<<<gridSize, blockSize>>>(a_d);

					    hipMemcpy(a_h, a_d, N * sizeof(int), hipMemcpyDeviceToHost);

					        // Print the results (optional) -  can also be removed if not needed
					        for (int i = 0; i < N; i++) {
							        printf("a[%d] = %d\n", i, a_h[i]);
								    }

						    hipFree(a_d);
						        free(a_h);

							    return 0;
}
