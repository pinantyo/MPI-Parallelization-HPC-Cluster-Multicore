
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* a) {
	    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	        a[idx] = threadIdx.x;  // Assign thread index to array element
}

int main() {
	    int N = 64;
	        int* a_h;
		    int* a_d;

		        a_h = (int*)malloc(N * sizeof(int));

			    hipMalloc((void**)&a_d, N * sizeof(int));

			        int blockSize = 16; 
				    int gridSize = (N + blockSize - 1) / blockSize;

				        kernel<<<gridSize, blockSize>>>(a_d);


					    hipMemcpy(a_h, a_d, N * sizeof(int), hipMemcpyDeviceToHost);


					        for (int i = 0; i < N; i++) {
							        printf("%d", a_h[i]);
								if ((i + 1) % blockSize == 0) {
									printf("\n");
								}
						}
						printf("\n");
								
						hipFree(a_d);
						free(a_h);

					        return 0;
}
