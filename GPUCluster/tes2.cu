
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* a) {
	    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	        a[idx] = blockIdx.x; // Assign block index to array element
}

int main() {
	    int N = 64;
	        int* a_h;
		    int* a_d;

		        a_h = (int*)malloc(N * sizeof(int));

			    hipMalloc((void**)&a_d, N * sizeof(int));

			        int blockSize = 16; //  jumlah thread disetiap blok
				    int gridSize = (N + blockSize - 1) / blockSize; // Calculate grid size

				        kernel<<<gridSize, blockSize>>>(a_d);


					    hipMemcpy(a_h, a_d, N * sizeof(int), hipMemcpyDeviceToHost);

					        for (int i = 0; i < N; i++) {
							        printf("%d", a_h[i]);
								    }
						    printf("\n");


						        hipFree(a_d);
							    free(a_h);

							        return 0;
}
