#include <iostream>

#include "mse.h"
#include "train.h"
#include "../utils/utils.h"


void train_gpu(Sequential_GPU seq, float *inp, float *targ, int bs, int n_in, int n_epochs){
    MSE_GPU mse(bs);

    int sz_inp = bs*n_in;
    float *cp_inp, *out;
    hipMallocManaged(&cp_inp, sz_inp*sizeof(float));

    for (int i=0; i<n_epochs; i++){
        set_eq(cp_inp, inp, sz_inp);

        seq.forward(cp_inp, out);
        mse.forward(seq.layers.back()->out, targ);

        mse.backward();
        seq.update();
    }

    seq.forward(inp, out);
    mse._forward(seq.layers.back()->out, targ);
    std::cout << "The final loss is: " << targ[bs] << std::endl;
}
